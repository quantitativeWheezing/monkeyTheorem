#include "hip/hip_runtime.h"
/* Kernels for random number generation and target matching */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// includes, project
#include "hip/hip_runtime_api.h"
#include "config.h"
#include "structs.h"
#include "monkeys_kernels.cuh"

//----------------------------------------------------------------------------//
//! Initialize hiprand
//! @param  state  to be initialized by hiprand
//! @param  seed  change seed for hiprand
//! @param  arrLen  how many numbers are generated and queried
//----------------------------------------------------------------------------//
__global__ void kernInitRand(hiprandState * __restrict__ state,
    const unsigned int seed,
    const size_t arrLen)
{

  const unsigned int tid = threadIdx.x+blockDim.x*blockIdx.x;
  if (tid<arrLen) {
    if (seed) {
      hiprand_init(tid+(unsigned int)clock(), tid, 0, &state[tid]);
    }
    else {
      hiprand_init(seed, tid, 0, &state[tid]);
    }
  }
}

//----------------------------------------------------------------------------//
//! Fill an array with random numbers: use this to test rng distribution
//! @param  state  to be initialized by hiprand
//! @param  arrLen  how many numbers are generated and queried
//! @param  typewriterSize  number of characters in alphabet used
//! @param  d_testDistInts  array to contain random numbers
//----------------------------------------------------------------------------//
__global__ void kernGenOnly(hiprandState * __restrict__ state, 
    const size_t arrLen, 
    const size_t typewriterSize, 
    unsigned int * __restrict__ d_testDistInts)
{

  const unsigned int tid = threadIdx.x+blockDim.x*blockIdx.x;
  if (tid < arrLen) {

    // generate random uint
    float fRoll = hiprand_uniform(&state[tid])*typewriterSize;
    unsigned int uiRoll = (unsigned int)trunc(fRoll);
    __syncthreads();
    d_testDistInts[tid] = uiRoll;
  }
}

//----------------------------------------------------------------------------//
//! Randomly generate numbers and compare element-wise matches to target
//! @param  state  must be initialized by hiprand
//! @param  seed  change seed for hiprand
//! @param  arrLen  how many numbers are generated and queried
//! @param  typewriterSize  number of characters in alphabet used
//! @param  d_targetInt  integer representation of target string
//! @param  targetLen  length of target string
//! @param  d_charMatch  indicates character-wise matches to target
//----------------------------------------------------------------------------//
__global__ void kernGenComp(hiprandState * __restrict__ state, 
    const size_t arrLen, 
    const size_t typewriterSize, 
    const unsigned int*__restrict__ d_targetInt,
    const unsigned int targetLen,
    bool * __restrict__ d_charMatch)
{

  const unsigned int tid = threadIdx.x+blockDim.x*blockIdx.x;

  // load target into shared memory
  volatile __shared__ unsigned int s_targetInt[SHMEM_SIZE+1];
  if ((threadIdx.x<targetLen)&&(tid<arrLen)) {
    s_targetInt[threadIdx.x] = d_targetInt[threadIdx.x];

    // duplicate target in shared mem to avoid bank conflicts later
    for(unsigned int i = threadIdx.x+BLOCK_SIZE_X; i<SHMEM_SIZE; 
        i += BLOCK_SIZE_X) {
      s_targetInt[i] = s_targetInt[threadIdx.x];
    }

  }
  __syncthreads();

  // generate random uint and compare to target
  bool match;
  if (tid<arrLen) {
    float fRoll = hiprand_uniform(&state[tid])*typewriterSize;
    unsigned int uiRoll = (unsigned int)trunc(fRoll);
    match = s_targetInt[threadIdx.x] == uiRoll;
    __syncthreads();
    d_charMatch[tid] = match;
  }
}

//----------------------------------------------------------------------------//
//! Vectorized memory access to check for contiguous matches: 4 char blocks
//! @param  arrLen  how many numbers are generated and queried
//! @param  targetLen  length of target string
//! @param  d_fullMatch  keeps track of when we find contiguous matches
//! @param  targetNum  consecutive "true" values represented as a number
//----------------------------------------------------------------------------//
#if TARGET_LENGTH == 4
  __global__ void kernVec4Match(const bool * __restrict__ d_charMatch,
      const size_t arrLen,
      const unsigned int targetLen,
      bool * __restrict__ d_fullMatch,
      const long targetNum)
  {

    const int tid = threadIdx.x+blockIdx.x*blockDim.x;

#if MATCH_VEC_SIZE == 4
    if (targetLen*4*tid<arrLen) {
      uint4 testNum;
      struct bool4 testBool;
      memcpy(&testNum, &d_charMatch[targetLen*4*tid], targetLen*4*sizeof(bool));
      testBool.x = testNum.x == targetNum;
      testBool.y = testNum.y == targetNum;
      testBool.z = testNum.z == targetNum;
      testBool.w = testNum.w == targetNum;
      __syncthreads();
      memcpy(&d_fullMatch[4*tid], &testBool, 4*sizeof(bool));
    }

#elif MATCH_VEC_SIZE == 2
    if (targetLen*2*tid<arrLen) {
      uint2 testNum;
      struct bool2 testBool;
      memcpy(&testNum, &d_charMatch[targetLen*2*tid], targetLen*2*sizeof(bool));
      testBool.x = testNum.x == targetNum;
      testBool.y = testNum.y == targetNum;
      __syncthreads();
      memcpy(&d_fullMatch[2*tid], &testBool, 2*sizeof(bool));
    }
#endif // #if MATCH_VEC_SIZE is 2 xor 4
  }

//----------------------------------------------------------------------------//
//! Vectorized memory access to check for contiguous matches: 8 char blocks
//! @param  arrLen  how many numbers are generated and queried
//! @param  targetLen  length of target string
//! @param  d_fullMatch  keeps track of when we find contiguous matches
//! @param  targetNum  consecutive "true" values represented as a number
//----------------------------------------------------------------------------//
#elif TARGET_LENGTH == 8
  __global__ void kernVec8Match(const bool * __restrict__ d_charMatch,
      const size_t arrLen,
      const unsigned int targetLen,
      bool * __restrict__ d_fullMatch,
      const long targetNum)
  {

    const int tid = threadIdx.x+blockIdx.x*blockDim.x;

#if MATCH_VEC_SIZE == 4
    if (targetLen*4*(tid+1)<arrLen) {
      ulong4 testNum;
      struct bool4 testBool;
      memcpy(&testNum, &d_charMatch[targetLen*4*tid], targetLen*4*sizeof(bool));
      testBool.x = testNum.x == targetNum;
      testBool.y = testNum.y == targetNum;
      testBool.z = testNum.z == targetNum;
      testBool.w = testNum.w == targetNum;
      __syncthreads();
      memcpy(&d_fullMatch[4*tid], &testBool, 4*sizeof(bool));
    }

#elif MATCH_VEC_SIZE == 2
    if (targetLen*2*(tid+1)<arrLen) {
      ulong2 testNum;
      struct bool2 testBool;
      memcpy(&testNum, &d_charMatch[targetLen*2*tid], targetLen*2*sizeof(bool));
      testBool.x = testNum.x == targetNum;
      testBool.y = testNum.y == targetNum;
      __syncthreads();
      memcpy(&d_fullMatch[2*tid], &testBool, 2*sizeof(bool));
    }
#endif // #if MATCH_VEC_SIZE is 2 xor 4
  }

#endif // #if TARGET_LENGTH is 4 xor 8
